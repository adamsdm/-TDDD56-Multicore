#include "hip/hip_runtime.h"
// Mandelbrot explorer, based on my old Julia demo plus parts of Nicolas Melot's Lab 1 code.
// CPU only! Your task: Rewrite for CUDA! Test and evaluate performance.

// Compile with:
// gcc interactiveMandelbrot.cpp -shared-libgcc -lstdc++-static  -o interactiveMandelbrot -lglut -lGL
// or
// g++ interactiveMandelbrot.cpp -o interactiveMandelbrot -lglut -lGL

// Your CUDA version should compile with something like
// nvcc -lglut -lGL interactiveMandelbrotCUDA.cu -o interactiveMandelbrotCUDA

// Preliminary version 2014-11-30
// Cleaned a bit more 2014-12-01
// Corrected the missing glRasterPos2i 2014-12-03




//nvcc interactiveMandelbrot_gpu.cu -o interactiveMandelbrot -lglut -lGL && ./interactiveMandelbrot



#ifdef __APPLE__
#include <OpenGL/gl.h>
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#include <GL/gl.h>
#endif
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

// Image data
	unsigned char	*pixels = NULL;
	int	 gImageWidth, gImageHeight;

// Init image data
void initBitmap(int width, int height)
{
	if (pixels) free(pixels);
	pixels = (unsigned char *)malloc(width * height * 4);
	gImageWidth = width;
	gImageHeight = height;
}

#define DIM 512

// Select precision here! float or double!
#define MYFLOAT float

// User controlled parameters
int maxiter = 5000;
MYFLOAT offsetx = -200, offsety = 0, zoom = 0;
MYFLOAT scale = 1.5;


// Complex number class
__device__ struct hipComplex
{
    MYFLOAT   r;
    MYFLOAT   i;

    __device__ hipComplex( MYFLOAT a, MYFLOAT b ) : r(a), i(b)  {}

    __device__ float magnitude2( void )
    {
        return r * r + i * i;
    }

    __device__ hipComplex operator*(const hipComplex& a)
    {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }

    __device__ hipComplex operator+(const hipComplex& a)
    {
        return hipComplex(r+a.r, i+a.i);
    }
};




__device__ int mandelbrot( int x, int y, int _gImageWidth, int _gImageHeight, MYFLOAT _offsetx, MYFLOAT _offsety , MYFLOAT _scale, int _maxiter)
{
    MYFLOAT jx = _scale * (MYFLOAT)(_gImageWidth/2 - x + _offsetx/_scale)/(_gImageWidth/2);
    MYFLOAT jy = _scale * (MYFLOAT)(_gImageHeight/2 - y + _offsety/_scale)/(_gImageWidth/2);

    hipComplex c(jx, jy);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<_maxiter; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return i;
    }

    return i;
}



char print_help = 0;

// Yuck, GLUT text is old junk that should be avoided... but it will have to do
static void print_str(void *font, const char *string)
{
	int i;

	for (i = 0; string[i]; i++)
		glutBitmapCharacter(font, string[i]);
}

void PrintHelp()
{
	if (print_help)
	{
		glPushMatrix();
		glLoadIdentity();
		glOrtho(-0.5, 639.5, -0.5, 479.5, -1.0, 1.0);

		glEnable(GL_BLEND);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glColor4f(0.f, 0.f, 0.5f, 0.5f);
		glRecti(40, 40, 600, 440);

		glColor3f(1.f, 1.f, 1.f);
		glRasterPos2i(300, 420);
		print_str(GLUT_BITMAP_HELVETICA_18, "Help");

		glRasterPos2i(60, 390);
		print_str(GLUT_BITMAP_HELVETICA_18, "h - Toggle Help");
		glRasterPos2i(60, 300);
		print_str(GLUT_BITMAP_HELVETICA_18, "Left click + drag - move picture");
		glRasterPos2i(60, 270);
		print_str(GLUT_BITMAP_HELVETICA_18,
		    "Right click + drag up/down - unzoom/zoom");
		glRasterPos2i(60, 240);
		print_str(GLUT_BITMAP_HELVETICA_18, "+ - Increase max. iterations by 32");
		glRasterPos2i(60, 210);
		print_str(GLUT_BITMAP_HELVETICA_18, "- - Decrease max. iterations by 32");
		glRasterPos2i(0, 0);

		glDisable(GL_BLEND);

		glPopMatrix();
	}
}

__global__ void computeFractalGPU(unsigned char *ptr, int _gImageHeight, int _gImageWidth, MYFLOAT _offsetx, MYFLOAT _offsety, MYFLOAT _scale, int _maxiter) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;




	int offset = x + y * _gImageWidth;

	// now calculate the value at that position
	int fractalValue = mandelbrot( x, y, _gImageWidth, _gImageHeight, _offsetx, _offsety , _scale, _maxiter);

	/*
	ptr[offset*4 + 0] = 255;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;

	ptr[offset*4 + 3] = 255;
	*/

	// Colorize it
	int red = 255 * fractalValue/_maxiter;
	if (red > 255) red = 255 - red;
	int green = 255 * fractalValue*4/_maxiter;
	if (green > 255) green = 255 - green;
	int blue = 255 * fractalValue*20/_maxiter;
	if (blue > 255) blue = 255 - blue;

	ptr[offset*4 + 0] = red;
	ptr[offset*4 + 1] = green;
	ptr[offset*4 + 2] = blue;

	ptr[offset*4 + 3] = 255;


}


float gpuTime;

// Compute fractal and display image
void Draw()
{
	const int size = gImageHeight * gImageWidth * 4 * sizeof(unsigned char);

	unsigned char	*d_pixels;
	hipMalloc((void**)&d_pixels, size);

	hipMemcpy(d_pixels, pixels, size, hipMemcpyHostToDevice);

	dim3 numBlocks(16, 16);
	dim3 threadsPerBlock(32, 32);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Execute the kernel
	hipEventRecord(start);
	computeFractalGPU <<<numBlocks, threadsPerBlock >>> (d_pixels, gImageHeight, gImageWidth, offsetx, offsety, scale, maxiter);
	hipDeviceSynchronize();
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	printf("%f \n", gpuTime);


	hipMemcpy(pixels, d_pixels, size, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();




// Dump the whole picture onto the screen. (Old-style OpenGL but without lots of geometry that doesn't matter so much.)
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );
	glDrawPixels( gImageWidth, gImageHeight, GL_RGBA, GL_UNSIGNED_BYTE, pixels );

	if (print_help)
		PrintHelp();

	glutSwapBuffers();
}

char explore = 1;

static void Reshape(int width, int height)
{
	glViewport(0, 0, width, height);
	glLoadIdentity();
	glOrtho(-0.5f, width - 0.5f, -0.5f, height - 0.5f, -1.f, 1.f);
	initBitmap(width, height);

	glutPostRedisplay();
}

int mouse_x, mouse_y, mouse_btn;

// Mouse down
static void mouse_button(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		// Record start position
		mouse_x = x;
		mouse_y = y;
		mouse_btn = button;
	}
}

// Drag mouse
static void mouse_motion(int x, int y)
{
	if (mouse_btn == 0)
	// Ordinary mouse button - move
	{
		offsetx += (x - mouse_x)*scale;
		mouse_x = x;
		offsety += (mouse_y - y)*scale;
		mouse_y = y;

		glutPostRedisplay();
	}
	else
	// Alt mouse button - scale
	{
		scale *= pow(1.1, y - mouse_y);
		mouse_y = y;
		glutPostRedisplay();
	}
}

void KeyboardProc(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 27: /* Escape key */
	case 'q':
	case 'Q':
		exit(0);
		break;
	case '+':
		maxiter += maxiter < 1024 - 32 ? 32 : 0;
		break;
	case '-':
		maxiter -= maxiter > 0 + 32 ? 32 : 0;
		break;
	case 'h':
		print_help = !print_help;
		break;
	}
	glutPostRedisplay();
}

// Main program, inits
int main( int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
	glutInitWindowSize( DIM, DIM );
	glutCreateWindow("Mandelbrot explorer (CPU)");
	glutDisplayFunc(Draw);
	glutMouseFunc(mouse_button);
	glutMotionFunc(mouse_motion);
	glutKeyboardFunc(KeyboardProc);
	glutReshapeFunc(Reshape);

	initBitmap(DIM, DIM);

	glutMainLoop();
}
