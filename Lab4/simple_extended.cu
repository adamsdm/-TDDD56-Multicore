
#include "hip/hip_runtime.h"


#include <stdio.h>

// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>

const int N = 16;
const int blocksize = 16;

__global__
void simple(float *a, float *b)
{
	b[threadIdx.x] = a[threadIdx.x] * a[threadIdx.x];
}

int main()
{
	float *a = new float[N];
	float *b = new float[N];
	float *da;
	float *db;
	const int size = N * sizeof(float);

	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, size);

	// Initialize data
	for (int i = 0; i < N; i++) {
		a[i] = i;
	}
	
	hipMemcpy(da, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, N * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	simple <<<dimGrid, dimBlock >>> (da, db);

	hipMemcpy(b, db, N * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		printf("%.2f^2 = %.2f\n", a[i], b[i]);
	}

	
	return EXIT_SUCCESS;
}
